
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>

#define BLOCK_SIZE 4

void FillMatrix(float* matrix, int height, int width) {
        srand(time(NULL));
        for (int i = 0; i < height; ++i) {
                for (int j = 0; j < width; ++j) {
                        matrix[i * width + j] = rand() % 2;
                }
        }
}

void Transpose(float *matrix, int height, int width) {
        float transposed[height * width];
        for (int i = 0; i < width; ++i) {
                for (int j = 0; j < height; ++j) {
                        transposed[i * height + j] = matrix[j * width + i];
                }
        }
        for (int i = 0; i < width * height; ++i) {
                matrix[i] = transposed[i];
        }
}

bool AreEqual(float* lhs_matrix, float* rhs_matrix, int height, int width) {
        for (int i = 0; i < height; ++i) {
                for (int j = 0; j < width; ++j) {
                        if (lhs_matrix[i * width + j] != rhs_matrix[i * width + j]) {
                                return false;
                        }
                }
        }
        return true;
}

__global__
void NaiveMatrixMul(float* A, float* B, float* C, int mid_size) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
 
        int width = blockDim.y * gridDim.y;

        C[i * width + j] = .0f;

        for (int k = 0; k < mid_size; ++k) {
                C[i * width + j] += A[i * mid_size + k] * B[k * width + j];
        }
}

__global__
void MatrixMul(float* A, float* B, float* C, int mid_size) {
        int width = blockDim.y * gridDim.y;

        int local_trow = threadIdx.x;
        int local_tcol = threadIdx.y;

        int block_row = blockIdx.x;
        int block_col = blockIdx.y;

        float tres = .0f;

        for (int k = 0; k < (mid_size / BLOCK_SIZE); ++k) {
                __shared__ float A_block[BLOCK_SIZE][BLOCK_SIZE];
                __shared__ float B_block[BLOCK_SIZE][BLOCK_SIZE];
                int A_block_idx = block_row * mid_size * BLOCK_SIZE + k * BLOCK_SIZE;
                int B_block_idx = block_col * mid_size * BLOCK_SIZE + k * BLOCK_SIZE;

                A_block[local_trow][local_tcol] = A[A_block_idx + local_trow * mid_size + local_tcol];
                B_block[local_tcol][local_trow] = B[B_block_idx + local_tcol * mid_size + local_trow];

                __syncthreads();

                for (int i = 0; i < BLOCK_SIZE; ++i) {
                        tres += A_block[local_trow][i] * B_block[local_tcol][i];
                }
                __syncthreads();
        }
        C[block_row * BLOCK_SIZE * width + local_trow * width + block_col * BLOCK_SIZE + local_tcol] = tres;
}

int main() {
        float *h_A;
        float *h_B;
        float *h_C;
        float *h_C_naive;

        int A_height, A_width, B_height, B_width;

        std::cin >> A_height >> A_width >> B_height >> B_width;

        h_A = new float[A_height * A_width];
        h_B = new float[B_height * B_width];
        h_C = new float[A_height * B_width];
        h_C_naive = new float[A_height * B_width];

        FillMatrix(h_A, A_height, A_width);
        FillMatrix(h_B, B_height, B_width);

        float* d_A;
        float* d_B;
        float* d_C;

        hipMalloc(&d_A, sizeof(float) * A_height * A_width);
        hipMalloc(&d_B, sizeof(float) * B_height * B_width);
        hipMalloc(&d_C, sizeof(float) * A_height * B_width);

        hipMemcpy(d_A, h_A, sizeof(float) * A_height * A_width, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, sizeof(float) * B_height * B_width, hipMemcpyHostToDevice);

        dim3 num_blocks(A_height / BLOCK_SIZE, B_width / BLOCK_SIZE);
        dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);

        hipEvent_t naive_start;
        hipEvent_t naive_stop;
        hipEventCreate(&naive_start);
        hipEventCreate(&naive_stop);

        hipEventRecord(naive_start);
        NaiveMatrixMul<<<num_blocks, block_size>>>(d_A, d_B, d_C, A_width);
        hipEventRecord(naive_stop);
        hipMemcpy(h_C_naive, d_C, sizeof(float) * A_height * B_width, hipMemcpyDeviceToHost);
        hipEventSynchronize(naive_stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, naive_start, naive_stop);
        std::cout << "Duration of naive method:" << milliseconds << "\n";

        hipEvent_t start;
        hipEvent_t stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        Transpose(h_B, B_height, B_width);
        hipMemcpy(d_B, h_B, sizeof(float) * B_height * B_width, hipMemcpyHostToDevice);
        hipEventRecord(start);
        MatrixMul<<<num_blocks, block_size>>>(d_A, d_B, d_C, A_width);
        hipEventRecord(stop);
        hipMemcpy(h_C, d_C, sizeof(float) * A_height * B_width, hipMemcpyDeviceToHost);
        hipEventSynchronize(stop);

        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout << "Duration of not naive method:" << milliseconds << "\n";

        if (AreEqual(h_C, h_C_naive, A_height, B_width)) {
                std::cout << "Matrices are equal\n";
        } else {
                std::cout << "Matrices are not equal\n";
        }

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        delete[] h_A;
        delete[] h_B;
        delete[] h_C;
        delete[] h_C_naive;

        return 0;
}
